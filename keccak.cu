
#include <hip/hip_runtime.h>
/*
 * keccak.cu  Implementation of Keccak/SHA3 digest
 *
 * Date: 12 June 2019
 * Revision: 1
 *
 * This file is released into the Public Domain.
 */
 
// Edited & optimized by krlnokrl
 
 
typedef unsigned char BYTE;
typedef uint32_t  WORD;
typedef uint64_t LONG; 
 

#define KECCAK_ROUND 24
#define KECCAK_STATE_SIZE 25
#define KECCAK_Q_SIZE 65

#define DIGESTBITLEN 256
#define RATE_BITS 1088 	//1600 - (256 << 1)
#define RATE_BYTES 136 	// RATE_BITS >> 3
#define ABSORB_ROUND 17	//RATE_BITS >> 6

__constant__ LONG CUDA_KECCAK_CONSTS[24] = { 0x0000000000000001, 0x0000000000008082,
                                          0x800000000000808a, 0x8000000080008000, 0x000000000000808b, 0x0000000080000001, 0x8000000080008081,
                                          0x8000000000008009, 0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
                                          0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003, 0x8000000000008002,
                                          0x8000000000000080, 0x000000000000800a, 0x800000008000000a, 0x8000000080008081, 0x8000000000008080,
                                          0x0000000080000001, 0x8000000080008008 };

typedef struct {

    int64_t state[KECCAK_STATE_SIZE];
    BYTE q[KECCAK_Q_SIZE];

    WORD bits_in_queue;

} cuda_keccak_ctx_t;
typedef cuda_keccak_ctx_t CUDA_KECCAK_CTX;


__device__ __forceinline__ static uint64_t xor5(const uint64_t a, const uint64_t b, const uint64_t c, const uint64_t d, const uint64_t e)
{
	uint64_t result;
	asm("xor.b64 %0, %1, %2;" : "=l"(result) : "l"(d) ,"l"(e));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(c));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(b));
	asm("xor.b64 %0, %0, %1;" : "+l"(result) : "l"(a));
	return result;
}


__device__ __forceinline__ uint64_t cuda_keccak_ROTL64(const uint64_t x, const int offset) {
	uint64_t res;
	asm("{ // ROTL64 \n\t"
		".reg .u32 tl,th,vl,vh;\n\t"
		".reg .pred p;\n\t"
		"mov.b64 {tl,th}, %1;\n\t"
		"shf.l.wrap.b32 vl, tl, th, %2;\n\t"
		"shf.l.wrap.b32 vh, th, tl, %2;\n\t"
		"setp.lt.u32 p, %2, 32;\n\t"
		"@!p mov.b64 %0, {vl,vh};\n\t"
		"@p  mov.b64 %0, {vh,vl};\n\t"
	"}\n" : "=l"(res) : "l"(x) , "r"(offset)
	);
	return res;
}
/*__device__ __forceinline__ LONG cuda_keccak_ROTL64(LONG a, const int  b)
{
    return (a << b) | (a >> (64 - b));
}
*/



__device__ __forceinline__ static void cuda_keccak_permutations(cuda_keccak_ctx_t * ctx)
{

    int64_t* A = ctx->state;

    int64_t *a00 = A, *a01 = A + 1, *a02 = A + 2, *a03 = A + 3, *a04 = A + 4;
    int64_t *a05 = A + 5, *a06 = A + 6, *a07 = A + 7, *a08 = A + 8, *a09 = A + 9;
    int64_t *a10 = A + 10, *a11 = A + 11, *a12 = A + 12, *a13 = A + 13, *a14 = A + 14;
    int64_t *a15 = A + 15, *a16 = A + 16, *a17 = A + 17, *a18 = A + 18, *a19 = A + 19;
    int64_t *a20 = A + 20, *a21 = A + 21, *a22 = A + 22, *a23 = A + 23, *a24 = A + 24;
	
	int64_t c0;
	int64_t c1;
	int64_t c2;
	int64_t c3;
	int64_t c4;
	
	int64_t d0;
	int64_t d1;
	int64_t d2;
	int64_t d3;
	int64_t d4;
	
	#pragma unroll 2
    for (int i = 0; i < KECCAK_ROUND; i++) {

        /* Theta */
        /*
		c0 = *a00 ^ *a05 ^ *a10 ^ *a15 ^ *a20;
        c1 = *a01 ^ *a06 ^ *a11 ^ *a16 ^ *a21;
        c2 = *a02 ^ *a07 ^ *a12 ^ *a17 ^ *a22;
        c3 = *a03 ^ *a08 ^ *a13 ^ *a18 ^ *a23;
        c4 = *a04 ^ *a09 ^ *a14 ^ *a19 ^ *a24;
		*/
		c0 = xor5(*a00, *a05, *a10, *a15, *a20);
		c1 = xor5(*a01, *a06, *a11, *a16, *a21);
		c2 = xor5(*a02, *a07, *a12, *a17, *a22);
		c3 = xor5(*a03, *a08, *a13, *a18, *a23);
		c4 = xor5(*a04, *a09, *a14, *a19, *a24);
		
        d1 = cuda_keccak_ROTL64(c1, 1) ^ c4;
        d2 = cuda_keccak_ROTL64(c2, 1) ^ c0;
        d3 = cuda_keccak_ROTL64(c3, 1) ^ c1;
        d4 = cuda_keccak_ROTL64(c4, 1) ^ c2;
        d0 = cuda_keccak_ROTL64(c0, 1) ^ c3;

        *a00 ^= d1;
        *a05 ^= d1;
        *a10 ^= d1;
        *a15 ^= d1;
        *a20 ^= d1;
        *a01 ^= d2;
        *a06 ^= d2;
        *a11 ^= d2;
        *a16 ^= d2;
        *a21 ^= d2;
        *a02 ^= d3;
        *a07 ^= d3;
        *a12 ^= d3;
        *a17 ^= d3;
        *a22 ^= d3;
        *a03 ^= d4;
        *a08 ^= d4;
        *a13 ^= d4;
        *a18 ^= d4;
        *a23 ^= d4;
        *a04 ^= d0;
        *a09 ^= d0;
        *a14 ^= d0;
        *a19 ^= d0;
        *a24 ^= d0;

        /* Rho pi */
        c1 = cuda_keccak_ROTL64(*a01, 1);
        *a01 = cuda_keccak_ROTL64(*a06, 44);
        *a06 = cuda_keccak_ROTL64(*a09, 20);
        *a09 = cuda_keccak_ROTL64(*a22, 61);
        *a22 = cuda_keccak_ROTL64(*a14, 39);
        *a14 = cuda_keccak_ROTL64(*a20, 18);
        *a20 = cuda_keccak_ROTL64(*a02, 62);
        *a02 = cuda_keccak_ROTL64(*a12, 43);
        *a12 = cuda_keccak_ROTL64(*a13, 25);
        *a13 = cuda_keccak_ROTL64(*a19, 8);
        *a19 = cuda_keccak_ROTL64(*a23, 56);
        *a23 = cuda_keccak_ROTL64(*a15, 41);
        *a15 = cuda_keccak_ROTL64(*a04, 27);
        *a04 = cuda_keccak_ROTL64(*a24, 14);
        *a24 = cuda_keccak_ROTL64(*a21, 2);
        *a21 = cuda_keccak_ROTL64(*a08, 55);
        *a08 = cuda_keccak_ROTL64(*a16, 45);
        *a16 = cuda_keccak_ROTL64(*a05, 36);
        *a05 = cuda_keccak_ROTL64(*a03, 28);
        *a03 = cuda_keccak_ROTL64(*a18, 21);
        *a18 = cuda_keccak_ROTL64(*a17, 15);
        *a17 = cuda_keccak_ROTL64(*a11, 10);
        *a11 = cuda_keccak_ROTL64(*a07, 6);
        *a07 = cuda_keccak_ROTL64(*a10, 3);
        *a10 = c1;

        /* Chi */
        c0 = *a00 ^ (~*a01 & *a02);
        c1 = *a01 ^ (~*a02 & *a03);
        *a02 ^= ~*a03 & *a04;
        *a03 ^= ~*a04 & *a00;
        *a04 ^= ~*a00 & *a01;
        *a00 = c0;
        *a01 = c1;

        c0 = *a05 ^ (~*a06 & *a07);
        c1 = *a06 ^ (~*a07 & *a08);
        *a07 ^= ~*a08 & *a09;
        *a08 ^= ~*a09 & *a05;
        *a09 ^= ~*a05 & *a06;
        *a05 = c0;
        *a06 = c1;

        c0 = *a10 ^ (~*a11 & *a12);
        c1 = *a11 ^ (~*a12 & *a13);
        *a12 ^= ~*a13 & *a14;
        *a13 ^= ~*a14 & *a10;
        *a14 ^= ~*a10 & *a11;
        *a10 = c0;
        *a11 = c1;

        c0 = *a15 ^ (~*a16 & *a17);
        c1 = *a16 ^ (~*a17 & *a18);
        *a17 ^= ~*a18 & *a19;
        *a18 ^= ~*a19 & *a15;
        *a19 ^= ~*a15 & *a16;
        *a15 = c0;
        *a16 = c1;

        c0 = *a20 ^ (~*a21 & *a22);
        c1 = *a21 ^ (~*a22 & *a23);
        *a22 ^= ~*a23 & *a24;
        *a23 ^= ~*a24 & *a20;
        *a24 ^= ~*a20 & *a21;
        *a20 = c0;
        *a21 = c1;

        /* Iota */
        *a00 ^= CUDA_KECCAK_CONSTS[i];
    }
}



__device__ __forceinline__ void cuda_keccak_pad(cuda_keccak_ctx_t *ctx)
{
    ctx->q[ctx->bits_in_queue >> 3] |= (1L << (ctx->bits_in_queue & 7));

    ++(ctx->bits_in_queue);


    LONG full = ctx->bits_in_queue >> 6;
    LONG partial = ctx->bits_in_queue & 63;

    LONG offset = 0;
	
	#pragma unroll 4
    for (int i = 0; i < full; ++i) {
        ctx->state[i] ^= *((uint64_t*)(ctx->q + offset));
        offset += 8;
    }

    if (partial > 0) {
        LONG mask = (1L << partial) - 1;
        ctx->state[full] ^= *((uint64_t*)(ctx->q + offset)) & mask;
    }

    ctx->state[(RATE_BITS - 1) >> 6] ^= 9223372036854775808ULL;/* 1 << 63 */

    cuda_keccak_permutations(ctx);

    ctx->bits_in_queue = RATE_BITS;
}


__device__ __forceinline__ void cuda_keccak_init(cuda_keccak_ctx_t *ctx)
{
    memset(ctx, 0, sizeof(cuda_keccak_ctx_t));

    ctx->bits_in_queue = 0;
}

__device__ __forceinline__ void cuda_keccak_update(cuda_keccak_ctx_t *ctx, BYTE* const in, const WORD inlen){
	int64_t BYTEs = ctx->bits_in_queue >> 3;
	memcpy(ctx->q + BYTEs, in, inlen);
	BYTEs += inlen;
	ctx->bits_in_queue = BYTEs << 3;
}


__device__ __forceinline__ void cuda_keccak_final(cuda_keccak_ctx_t *ctx, BYTE *out)
{
    cuda_keccak_pad(ctx);
    WORD i = 0;
    memcpy(out, ctx->state , 8);

}